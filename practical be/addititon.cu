#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

// CUDA kernel to add two vectors
__global__ void vectorAdd(float *A, float *B, float *C, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N = 10; 
    size_t size = N * sizeof(float);

    // Host memory
    float *A, *B, *C;

    // Device memory
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    // Allocate device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        A[i] = i + 1;
        B[i] = (i + 1) * 2;
    }

    // Copy host arrays to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Kernel launch configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Output results
    cout << "Vector A: ";
    for (int i = 0; i < N; i++) {
        cout << A[i] << " ";
    }
    cout << endl;

    cout << "Vector B: ";
    for (int i = 0; i < N; i++) {
        cout << B[i] << " ";
    }
    cout << endl;

    cout << "Calculations (A[i] + B[i]):" << endl;
    for (int i = 0; i < N; i++) {
        cout << "C[" << i << "] = " << A[i] << " + " << B[i] << " = " << C[i] << endl;
    }

    // Free memory
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
